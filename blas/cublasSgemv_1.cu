#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <time.h>

void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
            double dr = (double)rand();
            A[j * M + i] = (dr / rMax) * 100.0;
        }
    }

    *outA = A;
}

int main(int argc, char **argv)
{
    int i;
    float *A, *dA;
    float *X, *dX;
    float *Y, *dY;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;

    /*
    alpha = 3.0f;
    beta = 16.0f;
    */

    alpha = 1;
    beta = 1;

    /*
    * M = # of rows
    * N = # of columns
    */
    
    int M = 100;
    int N = 5;

    struct timespec startTime, endTime, sleepTime;

    clock_gettime(CLOCK_REALTIME, &startTime);
    sleepTime.tv_sec = 0;
    sleepTime.tv_nsec = 123;

    // Generate inputs
    srand(9384);
    generate_random_dense_matrix(M, N, &A);
    generate_random_vector(N, &X);
    generate_random_vector(M, &Y);

    hipblasCreate(&handle);
    hipMalloc((void **)&dA, sizeof(float) * M * N);
    hipMalloc((void **)&dX, sizeof(float) * N);
    hipMalloc((void **)&dY, sizeof(float) * M);
    hipblasSetVector(N, sizeof(float), X, 1, dX, 1);
    hipblasSetVector(M, sizeof(float), Y, 1, dY, 1);
    hipblasSetMatrix(M, N, sizeof(float), A, M, dA, M);

    hipblasSgemv(handle, HIPBLAS_OP_N, M, N, &alpha, dA, M, dX, 1, &beta, dY, 1);
    hipblasGetVector(M, sizeof(float), dY, 1, Y, 1);

    clock_gettime(CLOCK_REALTIME, &endTime);

    for (i = 0; i < 10; i++)
    {
        printf("%2.2f\n", Y[i]);
    }

    printf("...\n");

    printf("started = %10ld.%09ld\n", startTime.tv_sec, startTime.tv_nsec);
    printf("finished = %10ld.%09ld\n", endTime.tv_sec, endTime.tv_nsec);
    printf("elapsed time = ");
    if (endTime.tv_nsec < startTime.tv_nsec) {
    	 printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec - 1 ,endTime.tv_nsec + 1000000000 - startTime.tv_nsec);
    } else {
	 printf("%ld.%09ld", endTime.tv_sec - startTime.tv_sec ,endTime.tv_nsec - startTime.tv_nsec);
    }
    printf(" sec\n");

    free(A);
    free(X);
    free(Y);

    hipFree(dA);
    hipFree(dY);
    hipblasDestroy(handle);

    return 0;
}
